#include "hip/hip_runtime.h"
#include "BNSL_GPU.cuh"

int *valuesRange, *samplesValues;

int nodesNum, samplesNum;

int allParentSetNumPerNode;

double * dev_lsTable;

int* globalBestGraph, *globalBestOrder;
double globalBestScore;

int initTime, calcLocalScoreTime, searchTime;

void BNSL_init() {
	startWatch();
	readNodeInfo(&nodesNum, &valuesRange);
	readSamples(&samplesValues, &samplesNum, nodesNum);
	initTime = stopWatch();
}

void BNSL_calcLocalScore() {
	startWatch();

	allParentSetNumPerNode = 0;
	for (int i = 0; i <= CONSTRAINTS; i++) {
		allParentSetNumPerNode = allParentSetNumPerNode + C(i, nodesNum - 1);
	}

	int * dev_valuesRange;
	int * dev_samplesValues;
	int * dev_N;

	// calculate max different values number for all pair of child and parent set
	int valuesMaxNum = calcValuesMaxNum(valuesRange, nodesNum);

	CUDA_CHECK_RETURN(
			hipMalloc(&dev_lsTable,
					nodesNum * allParentSetNumPerNode * sizeof(double)),
			"hipMalloc failed: dev_lsTable.");
	CUDA_CHECK_RETURN(hipMalloc(&dev_valuesRange, nodesNum * sizeof(int)),
			"hipMalloc failed: dev_valuesRange.");
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_samplesValues, samplesNum * nodesNum * sizeof(int)),
			"hipMalloc failed: dev_samplesValues.");
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_N,
					valuesMaxNum * allParentSetNumPerNode * sizeof(int)),
			"dev_N hipMalloc failed.");

	CUDA_CHECK_RETURN(
			hipMemcpy(dev_valuesRange, valuesRange, nodesNum * sizeof(int),
					hipMemcpyHostToDevice),
			"hipMemcpy failed: valuesRange -> dev_valuesRange");
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_samplesValues, samplesValues,
					samplesNum * nodesNum * sizeof(int),
					hipMemcpyHostToDevice),
			"hipMemcpy failed: samplesValues -> dev_samplesValues");

	int threadNum = 256;
	int blockNum = (allParentSetNumPerNode - 1) / threadNum + 1;
//	calcAllLocalScore_kernel<<<blockNum, threadNum, nodesNum * sizeof(int)>>>(
//			dev_valuesRange, dev_samplesValues, dev_lsTable, dev_N, samplesNum,
//			nodesNum, allParentSetNumPerNode, valuesMaxNum);

	CUDA_CHECK_RETURN(hipFree(dev_valuesRange),
			"hipFree failed: dev_valuesRange.");
	CUDA_CHECK_RETURN(hipFree(dev_samplesValues),
			"hipFree failed: dev_samplesValues.");
	CUDA_CHECK_RETURN(hipFree(dev_N), "hipFree failed: dev_N.");

	free(valuesRange);
	free(samplesValues);
	calcLocalScoreTime = stopWatch();
}

void BNSL_search() {
	startWatch();

	int i, j, iter;
	int parentSetNumInOrder = 0;
	for (i = 0; i < nodesNum; i++) {
		for (j = 0; j <= CONSTRAINTS && j < i + 1; j++) {
			parentSetNumInOrder += C(j, i);
		}
	}

	int ordersNum = 128;

	int iterNum = ITER;

	srand((unsigned int) time(NULL));

	int seed = 1234;

	int * dev_newOrders;
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_newOrders, ordersNum * nodesNum * sizeof(int)),
			"hipMalloc failed: dev_newOrders.");

	int * newOrder = (int *) malloc(nodesNum * sizeof(int));
	CUDA_CHECK_RETURN(hipHostMalloc(&newOrder, nodesNum * sizeof(int)),
			"hipHostMalloc failed: newOrder.");

	randInitOrder(newOrder, nodesNum);

	double * dev_parentSetScore;
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_parentSetScore,
					ordersNum * parentSetNumInOrder * sizeof(double)),
			"hipMalloc failed: dev_parentSetScore.");

	double * dev_maxLocalScore;
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_maxLocalScore,
					ordersNum * nodesNum * sizeof(double)),
			"hipMalloc failed: dev_maxLocalScore.");

	double * dev_ordersScore, *ordersScore;
	CUDA_CHECK_RETURN(hipMalloc(&dev_ordersScore, ordersNum * sizeof(double)),
			"hipMalloc failed: dev_ordersScore.");
	CUDA_CHECK_RETURN(hipHostMalloc(&ordersScore, ordersNum * sizeof(double)),
			"hipHostMalloc failed: ordersScore.");

	double *dev_prob, *prob;
	CUDA_CHECK_RETURN(hipMalloc(&dev_prob, ordersNum * sizeof(double)),
			"hipMalloc failed: dev_prob.");
	CUDA_CHECK_RETURN(hipHostMalloc(&prob, ordersNum * sizeof(double)),
			"hipHostMalloc failed: prob.");

	int *dev_samples, *samples;
	CUDA_CHECK_RETURN(hipMalloc(&dev_samples, ordersNum * sizeof(int)),
			"hipMalloc failed: dev_samples.");
	CUDA_CHECK_RETURN(hipHostMalloc(&samples, ordersNum * sizeof(int)),
			"hipHostMalloc failed: samples.");

	globalBestOrder = (int *) malloc(nodesNum * sizeof(int));
	globalBestScore = -FLT_MAX;

	hiprandState *dev_curandState;
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_curandState, ordersNum * sizeof(hiprandState)),
			"hipMalloc failed: dev_curandState.");

	curandSetup_kernel<<<1, ordersNum>>>(dev_curandState, seed);
	CUDA_CHECK_RETURN(hipGetLastError(), "curandSetup_kernel launch failed.");

	calcCDFInit(ordersNum);

	for (iter = 1; iter <= iterNum; iter++) {
		printf("iter = %d:\n", iter);

		//calcGPUTimeStart("generateOrders_kernel: ");
		CUDA_CHECK_RETURN(
				hipMemcpy(dev_newOrders, newOrder, nodesNum * sizeof(int),
						hipMemcpyHostToDevice),
				"hipMemcpy failed: newOrder -> dev_newOrders.");
		generateOrders_kernel<<<1, 128, nodesNum * sizeof(int)>>>(dev_newOrders,
				dev_curandState, nodesNum, ordersNum);
		CUDA_CHECK_RETURN(hipGetLastError(),
				"generateOrders_kernel launch failed.");
		//calcGPUTimeEnd();

		//calcGPUTimeStart("calcOnePairPerThread_kernel: ");
		int threadNum = 128;
		int blockNum = (parentSetNumInOrder - 1) / threadNum + 1;
		dim3 gridDim(blockNum, ordersNum);
		calcOnePairPerThread_kernel<<<gridDim, threadNum>>>(dev_lsTable,
				dev_newOrders, dev_parentSetScore, nodesNum,
				allParentSetNumPerNode, parentSetNumInOrder);
		CUDA_CHECK_RETURN(hipGetLastError(),
				"calcOnePairPerThread_kernel launch failed.");
		//calcGPUTimeEnd();

		//calcGPUTimeStart("calcMaxParentSetScoreForEachNode_kernel: ");
		calcMaxParentSetScoreForEachNode_kernel<<<nodesNum, ordersNum>>>(
				dev_parentSetScore, dev_maxLocalScore, parentSetNumInOrder,
				nodesNum);
		CUDA_CHECK_RETURN(hipGetLastError(),
				"calcMaxLocalScoreForEachNode_kernel launch failed.");
		//calcGPUTimeEnd();

		calcAllOrdersScore_kernel<<<1, ordersNum>>>(dev_maxLocalScore,
				dev_ordersScore, nodesNum);
		CUDA_CHECK_RETURN(hipGetLastError(),
				"calcAllOrdersScore_kernel launch failed.");
		CUDA_CHECK_RETURN(
				hipMemcpy(ordersScore, dev_ordersScore,
						ordersNum * sizeof(double), hipMemcpyDeviceToHost),
				"hipMemcpy failed: dev_ordersScore -> ordersScore.");

		int maxId = calcCDF(ordersScore, prob);

		if (ordersScore[maxId] > globalBestScore) {
			CUDA_CHECK_RETURN(
					hipMemcpy(globalBestOrder,
							dev_newOrders + maxId * nodesNum,
							nodesNum * sizeof(int), hipMemcpyDeviceToHost),
					"hipMemcpy failed: dev_newOrders -> globalBestOrder");
			globalBestScore = ordersScore[maxId];
		}

		CUDA_CHECK_RETURN(
				hipMemcpy(dev_prob, prob, ordersNum * sizeof(double),
						hipMemcpyHostToDevice),
				"hipMemcpy failed: prob -> dev_prob.");
		sample_kernel<<<1, ordersNum, ordersNum * 8>>>(dev_prob, dev_samples,
				dev_curandState, ordersNum);
		CUDA_CHECK_RETURN(hipGetLastError(), "sample_kernel launch failed.");
		CUDA_CHECK_RETURN(
				hipMemcpy(samples, dev_samples, ordersNum * sizeof(int),
						hipMemcpyDeviceToHost),
				"hipMemcpy failed: dev_samples -> samples.");

		int r = rand() % ordersNum;
		CUDA_CHECK_RETURN(
				hipMemcpy(newOrder, dev_newOrders + samples[r] * nodesNum,
						nodesNum * sizeof(int), hipMemcpyDeviceToHost),
				"hipMemcpy failed: dev_newOrders -> newOrder");
	}

	CUDA_CHECK_RETURN(hipFree(dev_newOrders),
			"hipFree failed: dev_newOrders.");
	CUDA_CHECK_RETURN(hipFree(dev_parentSetScore),
			"hipFree failed: dev_parentSetScore.");
	CUDA_CHECK_RETURN(hipFree(dev_maxLocalScore),
			"hipFree failed: dev_maxLocalScore.");
	CUDA_CHECK_RETURN(hipFree(dev_ordersScore),
			"hipFree failed: dev_ordersScore.");
	CUDA_CHECK_RETURN(hipFree(dev_prob), "hipFree failed: dev_prob.");
	CUDA_CHECK_RETURN(hipFree(dev_samples), "hipFree failed: dev_samples.");
	CUDA_CHECK_RETURN(hipFree(dev_curandState),
			"hipFree failed: dev_curandState.");
	CUDA_CHECK_RETURN(hipHostFree(newOrder), "hipHostFree failed: newOrder.");
	CUDA_CHECK_RETURN(hipHostFree(ordersScore),
			"hipHostFree failed: ordersScore.");
	CUDA_CHECK_RETURN(hipHostFree(prob), "hipHostFree failed: prob.");
	CUDA_CHECK_RETURN(hipHostFree(samples), "hipHostFree failed: samples.");

	calcCDFFinish();

	searchTime = stopWatch();
}

void BNSL_printResult() {

	printf("Best Score: %f \n", globalBestScore);
	printf("Best Topology: ");
	for (int i = 0; i < nodesNum; i++) {
		printf("%d ", globalBestOrder[i]);
	}
	printf("\n");
	printf("BNSL_init elapsed time is %dms.\n", initTime);
	printf("BNSL_calcLocalScore time is %dms. \n", calcLocalScoreTime);
	printf("BNSL_search time is %dms. \n", searchTime);
}

void BNSL_finish() {
	CUDA_CHECK_RETURN(hipFree(dev_lsTable), "hipFree failed: dev_lsTable.");
	free(globalBestOrder);
	free(globalBestGraph);
}
